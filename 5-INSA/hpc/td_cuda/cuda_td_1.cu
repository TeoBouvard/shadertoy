#include "hip/hip_runtime.h"
#include "wb.hpp"
#include <stdlib.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    out[i] = in1[i] + in2[i];
  }
}

int main(int argc, char **argv) {
  // read arguments
  wbArg_t args = wbArg_read(argc, argv);

  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  // read input data
  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 = (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");
  wbLog(TRACE, "The input length is ", inputLength);

  // allocate GPU memory
  wbTime_start(GPU, "Allocating GPU memory.");
  hipMalloc(&deviceInput1, inputLength * sizeof(float));
  hipMalloc(&deviceInput2, inputLength * sizeof(float));
  hipMalloc(&deviceOutput, inputLength * sizeof(float));
  wbTime_stop(GPU, "Allocating GPU memory.");

  // copy memory to the GPU
  wbTime_start(GPU, "Copying input memory to the GPU.");
  hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  // initialize grid and block dimensions
  // hard to decide which execution configuration should be chosen
  int numBlocks = 1;
  int threadsPerBlock = inputLength;

  // launch GPU kernel
  wbTime_start(Compute, "Performing CUDA computation");
  vecAdd<<<numBlocks, threadsPerBlock>>>(deviceInput1, deviceInput2,
                                         deviceOutput, inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  // copy GPU memory back to the CPU
  wbTime_start(Copy, "Copying output memory to the CPU");
  hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  // free GPU memory
  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  // check solution
  wbSolution(args, hostOutput, inputLength);

  // free host memory
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return EXIT_SUCCESS;
}
